#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include ""
#include "E:\Matlab R2018a\extern\include\mex.h"

#define N   144 //��������
#define P   3  //��������
#define POP 512 //��Ⱥ���� 
#define SIZE N*P*POP 
#define ALPHA 100 //�ͷ�����1
#define BETA 100  //�ͷ�����2

//�������BLOCKS��HREAD_NUM 
#define BLOCKS POP
#define THREAD_NUM P //THREAD_NUM С�ڵ���POP����POP�ܳ���THREAD_NUM,THREAD_NUM���1024��THREAD_NUM�����32,һ�����ó�P*POP,���������1024������BLOCKS
//xתxt��HREAD_NUM����ʱ��ҪBLOCKS


#define PSO_THREAD_NUM 512 //����߳�����������ܳ���POP�����ó�POP
#define PSO_BLOCKS SIZE/PSO_THREAD_NUM 

#define RD_THREAD_NUM 512
#define RD_BLOCKS POP/RD_THREAD_NUM
#define W 1
#define C1 2
#define C2 2
#define STOPNUM 50

#define data "wine.txt"
#define U N/P
#define ITER 30
#define MAXITER 500
//cublas����˷��ھ���,A[m,k], B[k,n],C[n,m],alpha=1.0,beta=0.0
//C=A*B
//hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

//��ע
//���ڴ�Ƕȣ�c���Եľ����ǰ������У�cublas�ľ����ǰ�������
//x[n,p*pop],����n������������p�����pop����Ⱥ��С
//�����к�s1,�轫xת��xt[n*pop,p],I1[p,1],��������s1=xt*I1,s1[n*pop,1]
//�����к�s2,I2[1,n],s2=I2*x,s2[1,p*pop]
//����t=d*x,d[n,n],x[n,p*pop],t[p*pop,n]
//index[n,p*pop],�������¾���

void generate_rd_num(int* a)
{
	int i, j;

	int b[N];
	// int* b = (int*)malloc(sizeof(int) * N);
	for (i = 0; i < N; i++)
		a[i] = i;
	srand(time(NULL));
	int temp;
	for (i = 0; i < N; i++)
	{
		//j = (int)((float)((N - i) * rand()) / (RAND_MAX + 1.0));
		j = rand() % N;
		temp = a[i];
		a[i] = a[j];
		a[j] = temp;
	}
}

//�첽���£�����batch�����浽����index[n,p*pop]��index��ÿ���е���Ԫ����ͬ�����£�ÿ��ֻ���첽����
void generate_batch(int *index)
{
	int pos;
	int *block = (int*)malloc(sizeof(int) * N*P);
	int* rd = (int*)malloc(sizeof(int) * N);
	int i, j, loop;
	for (i = 0; i < N; i++)
		for (j = 0; j < P; j++)
			block[i*P + j] = (i*P + j * (P + 1)) % (N*P);

	//print_matrix_int(block, N, P);

	for (loop = 0; loop < POP; loop++)
	{
		generate_rd_num(rd);

		//for (int ii=0;ii<N;ii++)
		//	printf("%d ",rd[ii]);
		//	printf("\n");
		//	printf("\n");
		for (i = 0; i < N; i++)
			for (j = 0; j < P; j++)
			{
				index[i*P*POP + loop * P + j] = block[rd[i] * P + j];
			}
		//print_index(index, N, P,POP,loop);
	}
	free(rd);
	free(block);
}



//����ͬ������ 
__global__ static void parallel_updated_GPU(float *obj, float *lbest, float *lbestx, float*initialx, float *x, float *d, float *s1, float *s2, float *t, int *index, int gen, hiprandState *globalState)
{
	hiprandGenerator_t gen1;  //�������������
	int tid, i, k, b, j;
	int loop;
	//hiprandState state;
	//hiprand_init(seed, tid, 0, &state);
	tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	int rd[N], temp, ped;
	hiprandState localState = globalState[tid];
	if (tid%P == 0)
	{

		//������һ�г�ʼ����
		for (int i = 0; i < N; i++)
			rd[i] = i * P;
		//���ҵ�һ��

		for (int i = 0; i < N; i++)
		{


			j = (hiprand(&localState)) % N;
			temp = rd[i];
			rd[i] = rd[j];
			rd[j] = temp;
		}
		//���������࣬��಻������P���Ҵ���P
		while (1)
		{
			ped = (hiprand(&localState)) % N;
			if (ped%P != 0 && ped > P)
				break;
		}
		for (i = 0; i < N; i++)
			for (j = 0; j < P; j++)
				index[i*P*POP + tid + j] = (ped * j + rd[i]) % (N*P);
	}
	for (i = 0; i < N; i++)
	{
		x[i*P*POP + tid] = initialx[i*P*POP + tid];
	}
	__syncthreads();





	b = tid / P;
	float flag = 1.0;
	int it = 0;
	while (it < ITER && (flag != 0.0))
	{
		flag = 0;
		//float total=0.0,const1=0.0,const2=0.0;
		int tidtemp = (hiprand(&localState)) % (POP*P);
		for (loop = 0; loop < N; loop++)
		{

			temp = index[loop*POP*P + tid];
			i = temp / P;
			k = temp % P + b * P;

			//loopȡֵ0~N-1
			//int tid =  threadIdx.x;
			int pos = P * POP*i + k; //�����̺߳ż����index����
			float x_pos = x[pos];//��������ȡ��x�����ݣ�׼������x(pos)
			float half = 0.5;
			//float s1_i = s1[b*N + i];//����s1(i)
			float s1_i = s1[i*POP + b];//����s1(i)
			float s2_k = s2[k];//����s2(k)
			float dedx = t[pos] + ALPHA * (s1_i - x_pos - 2 + P) + BETA * (s2_k - x_pos - 2 * U + N);
			//float dedx=t[k*N+i]+ALPHA*(s1_i-half-x_pos)+BETA*(s2_k-x_pos-U+half);
			//float dedx=ALPHA*(s1_i-half-x_pos)+BETA*(s2_k-x_pos-U+half);

			if (-dedx > 0)
				x[pos] = 1.0;
			else if (-dedx < 0)
				x[pos] = -1.0;

			//��������Ż���if x[i][k]==0 t[k][j]=0
			//for(int j=0;j<N;j++)
				//t[k*N+j]=d[j*N+i]*x[pos];
			if (x[pos] != x_pos)
			{
				s1[i*POP + b] = s1_i - x_pos + x[pos];
				s2[k] = s2_k - x_pos + x[pos];
				float deltx = x[pos] - x_pos;
				for (int j = 0; j < N; j++)
					t[j*P*POP + k] = t[j*P*POP + k] + d[j*N + i] * deltx;
			}
			//__syncthreads();
			flag = flag + (s1[i*POP + b] - 2 + P)*(s1[i*POP + b] - 2 + P) + (s2[k] - 2 * U + N)*(s2[k] - 2 * U + N);
		}
		it++;
		//if (flag == 0)
		//	break;
	}
	__syncthreads();
	if (tid%P == 0)
	{
		float total = 0.0, const1 = 0.0, const2 = 0.0;
		int id;
		for (k = tid; k < P + tid; k++)
			for (i = 0; i < N; i++)
			{
				id = i * P*POP + k;
				if (x[id] == 1.0)
					total = total + t[id];
				else
					total = total - t[id];
			}
		for (i = 0; i < N; i++)
			const1 = const1 + ALPHA * (s1[i*POP + b] - 2 + P)*(s1[i*POP + b] - 2 + P);
		for (k = 0; k < P; k++)
			const2 = const2 + BETA * (s2[b*P + k] - 2 * U + N)*(s2[b*P + k] - 2 * U + N);
		total = total * 0.5 + const1 + const2;
		obj[tid / P] = total;
		//����ǵ�һ�ε�������ô������Ⱥ�õ���Ŀ�꺯��ֵ���϶������ŵ�
		if (gen == 0)
		{
			lbest[tid / P] = total;
			for (k = tid; k < tid + P; k++)
				for (i = 0; i < N; i++)
				{
					id = i * P*POP + k;
					lbestx[id] = x[id];
				}
		}
		else
		{
			if (obj[tid / P] < lbest[tid / P])
			{

				lbest[tid / P] = obj[tid / P];
				for (k = tid; k < tid + P; k++)
					for (i = 0; i < N; i++)
					{
						id = i * P*POP + k;
						lbestx[id] = x[id];
					}
			}
		}
	}



}


//���ļ���ȡd
void generate_d(float* d, char* str)
{
	FILE* fp;            /*�ļ�ָ��*/
	errno_t error;
	float* TemporaryD = d;
	error = fopen_s(&fp, str, "r");
	if (error != 0)
	{
		perror("fail to read");
		exit(1);
	}
	for (size_t i = 0; i < N; i++) {
		for (size_t j = 0; j < N; j++)
		{
			fscanf_s(fp, "%f", TemporaryD);
			TemporaryD++;
			fseek(fp, 1L, SEEK_CUR);   /*fpָ��ӵ�ǰλ������ƶ�*/
		}
	}
	fclose(fp);                     //�ر��ļ�
}
void generate_s1(hipblasHandle_t handle, float* s1, float *x, float* I1)
{

	//���Դ��е�xת����xt,һ��THREAD_NUM����POP�����һ���Կ����߳���������POP������forѭ������
	//for (int loop = 0; loop < REPEAT; loop++)
	//	trans_x_xt << <1, TRANS_THREAD_NUM >> > (xt, x, loop);


	//x[n,p*sizepop]
	//����s1,s1=x*I1,x���кͱ��浽s1

	const float alpha = 1.0f;
	const float beta = 0.0f;
	//�����к�s1,s1=xt*I1,I1[p,1],�൱��xt[n*pop,p]��A[m,k]��I1[p,1]��B[k,n]��
	//hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N*POP, P, &alpha, I1, 1, xt, P, &beta, s1, 1);
	//�����к�s1,s1=x*I1,I1[p,1],�൱��x[n,p*pop]��A[m,k]��I1[p*pop,pop]��B[k,n]��
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, POP, N, P*POP, &alpha, I1, POP, x, P*POP, &beta, s1, POP);
}

void generate_s2(hipblasHandle_t handle, float* s2, float* x, float* I2)
{
	//x[n,p*sizepop]
	//����s1,s1=x*I1,x���кͱ��浽s1


	const float alpha = 1.0f;
	const float beta = 0.0f;
	//����s2,s2=I2*x,x���кͱ��浽s2
	//�����к�s1,s2=I2*x,I2[1,n],�൱��I2[1,n]��A[m,k]��x[n,p*pop]��B[k,n]��
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, P*POP, 1, N, &alpha, x, P*POP, I2, N, &beta, s2, P*POP);

}

void generate_t(hipblasHandle_t handle, float* t, float* d, float* x)
{
	const float alpha = 1.0f;
	const float beta = 0.0f;
	//����t=d*x,d[n,n],x[n,p*pop],t[p*pop,n],�൱��d[n,n]��A[m,k]��x[n,p*pop]��B[k,n]��
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, P*POP, N, N, &alpha, x, P*POP, d, N, &beta, t, P*POP);

}

void generate_I1(float* I)
{
	int i, j;
	for (i = 0; i < POP; i++)
	{
		for (j = 0; j < P*POP; j++)
			I[j*POP + i] = 0.0;
		for (j = i * P; j < (i + 1)*P; j++)
			I[j*POP + i] = 1.0;
	}
}
void generate_I2(float* I, int num)
{
	int i;
	for (i = 0; i < num; i++)
		I[i] = 1.0;
}


void initial_x(float* x)
{
	int i;
	float rd;
	srand((unsigned)time(NULL));
	for (i = 0; i < SIZE; i++) {

		rd = 2.0 * rand() / RAND_MAX - 1; //����-1��1�������
		if (rd > 0)
			x[i] = 1.0;
		else
			x[i] = -1.0;
	}
}

void initial_obj(float* x)
{
	int i;
	for (i = 0; i < POP; i++)
		x[i] = 0.0;
}

//����ȫ������Ŀ�꺯��ֵ�����Ž�,
__global__ static void generate_global_best(float *y, float *gbest, float *gbestx, float * lbest, float *lbestx, int it)
{

	int i, j, k, flag = 0;
	//������Ⱥ0��Ŀ�꺯��ֵ��С�Ҿ������Ž�
	float gb = lbest[0];
	int id = 0;
	//��ȡȫ�����Ž�
	for (i = 0; i < POP; i++)
		if (lbest[i] < gb)
		{
			gb = lbest[i];
			id = i;
		}
	if (it == 0 || gb < *gbest)
	{
		*gbest = gb;
		y[it] = gb;
		k = 0;
		int count = 0;
		for (j = 0; j < N; j++)
			for (i = id * P; i < (id + 1)*P; i++)
				gbestx[k++] = lbestx[j*P*POP + i];

		/*
			for (j = 0; j < N; j++)
			{
				for (i = id * P; i < (id + 1)*P; i++)
					printf("%f ", lbestx[j*P*POP + i]);
				printf("\n");
			}*/
	}
	else
	{
		y[it] = *gbest;
	}

	//printf("%f\n", y[it]);


}
__global__ static void pso(float *x, float *initialx, float *initialv, float *gbestx, float *lbestx, float *rd1, float *rd2,unsigned long seed)
{

	int i, k;
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1


	//printf("%f %f\n", rd1,rd2);
	//float rd1 = 0.7;
	//float rd2 = 0.5;
	/*
	if (tid == 0)
	{
		printf("\n");
		for (i = 0; i < N; i++)
		{
			for (k = 0; k < P; k++)
				printf("% f", initialx[i*P + k]);
			printf("\n");
		}
	}*/
	i = tid / (P*POP);
	k = tid % (P*POP) % P;

	/*
	   initial_v{j}=w1*initial_v{j}+beta1*rd1*(pbest_x{j}-initial_x{j})+beta2*rd2*(zbest_x-initial_x{j});
	   initial_x{j}=initial_x{j}+initial_v{j};
	   initial_x{j}=round(min(1,max(0,initial_x{j})));  %for zero one
	*/

	initialv[tid] = W * initialv[tid] + C1 * rd1[tid % (P*POP) / P] * ((lbestx[tid]+1)/2.0 - (initialx[tid]+1)/2.0) + C2 * rd2[tid % (P*POP) / P] * ((gbestx[i*P + k]+1)/2.0 - (initialx[tid]+1)/2.0);
	/*
	initialx[tid] = initialx[tid] + initialv[tid];
	if (initialx[tid] < 0.0)
		initialx[tid] = -1.0;
	if (initialx[tid] >= 1.0)
		initialx[tid] = 1.0;
	if (initialx[tid] >= 0.5)
		initialx[tid] = 1.0;
	else
		initialx[tid] = -1.0;
		*/
	initialv[tid] = W * initialv[tid] + C1 * rd1[tid % (P*POP) / P] * (lbestx[tid] - initialx[tid]) + C2 * rd2[tid % (P*POP) / P] * (gbestx[i*P + k] - initialx[tid]);
	float s, rd;
	s = 1 / (1 + exp(-initialv[tid]));
	hiprandState state;
	hiprand_init(seed, tid, 0, &state);
	rd = hiprand_uniform(&state);
	if (s > rd)
		initialx[tid] = 1.0;
	else
		initialx[tid] = -1.0;

	/*
if(initialv[tid]>0.0)
	initialv[tid]=1.0;
else
	initialv[tid]=-1.0;
	*/
	/*
	if (tid == 0)
	{
		printf("\n");
		for (i = 0; i < N; i++)
		{
			for (k = 0; k < P; k++)
				printf("% f", initialx[i*P + k]);
			printf("\n");
		}
	}*/


}

void initial_v(float* x)
{
	int i;
	float rd;
	srand((unsigned)time(NULL));
	for (i = 0; i < SIZE; i++) {

		rd = 2.0 * rand() / RAND_MAX - 1; //����-1��1�������
		if (rd > 0)
			x[i] = 1.0;
		else
			x[i] = -1.0;
	}
}
__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	hiprand_init(seed, tid, 0, &state[tid]);// initialize the state
}
__global__ void generate_rd1_rd2(float *rd1, float *rd2, hiprandState *globalState)
{
	int tid = blockIdx.x *blockDim.x + threadIdx.x; //��ȡ�̺߳�0~blocks*THREAD_NUM-1
	hiprandState localState = globalState[tid];
	rd1[tid] = hiprand_uniform(&localState);
	rd2[tid] = hiprand_uniform(&localState);
}
__global__ void stop(float *y, int *is_stop, int it)
{
	*is_stop = 0;
	if (y[it] == y[it - STOPNUM])
		*is_stop = 1;
}

void Hopfield_syn_cuda(float *y,float *gbestx)
{
	//ѡ���Կ�,ubuntu�û���3��2080ti�Կ������ϴ�ѧ��2��rtx8000�Կ�
	hipSetDevice(1);

	//�������ʱ�䣬��ʱ����
	/*clock_t start, finish;
	double  duration;
	start = clock();
	finish = clock();
	duration = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("%f seconds\n", duration);
	*/

	//��ʼ��x
	//����x��xt���ڴ����Դ���� 
	float *xh, *xth, *xd, *xtd;
	xh = (float*)malloc(sizeof(float) * SIZE); //x���ڴ�����������ڴ�ռ�
	xth = (float*)malloc(sizeof(float)*SIZE); //xt���ڴ�����������ڴ�ռ�
	initial_x(xh);//printf("\n");print_matrix(xh, N, P*POP);//��ʼ��x
	hipMalloc((void**)&xd, sizeof(float) * SIZE); //x���Դ�����������Դ�ռ�
	hipMalloc((void**)&xtd, sizeof(float) * SIZE); //xtd���Դ�����������Դ�ռ�
	hipMemcpy(xd, xh, sizeof(int) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�

	//���Կ����㣬��ʱ����#pragma omp parallel for num_threads(3)


	//����x��xt�Ƿ���ȷ����ʱ����
	/*print_matrix(xh, N, P*POP);
	hipMemcpy(xth, xtd, sizeof(int)*SIZE, hipMemcpyDeviceToHost); //�Դ����ݿ������ڴ�
	print_matrix(xth, N* POP, P);*/

	//����obj,index,I1��I2��s1��s2��t��d���ڴ����Դ���� 
	int *indexd, *indexh;
	float *sd1, *sd2, *sh1, *sh2, *Id1, *Id2, *Ih1, *Ih2, *td, *th, *dd, *dh;
	float *objd, *objh;
	float *initialxh, *initialxd, *initialvh, *initialvd;
	initialxh = (float*)malloc(sizeof(float)*SIZE);
	initialvh = (float*)malloc(sizeof(float)*SIZE); initial_v(initialvh);
	hipMalloc((void**)&initialvd, sizeof(float) * SIZE);
	indexh = (int*)malloc(sizeof(int) *SIZE); generate_batch(indexh);//index���ڴ�����������ڴ�ռ�
	Ih1 = (float*)malloc(sizeof(float)  * POP*P*POP); generate_I1(Ih1);//I1���ڴ�����������ڴ�ռ�
	Ih2 = (float*)malloc(sizeof(float) * N); generate_I2(Ih2, N);//I2���ڴ�����������ڴ�ռ�
	sh1 = (float*)malloc(sizeof(float) * N*POP);//s1���ڴ�����������ڴ�ռ�
	sh2 = (float*)malloc(sizeof(float) * P*POP);//s2���ڴ�����������ڴ�ռ�
	th = (float*)malloc(sizeof(float) * SIZE);//t���ڴ�����������ڴ�ռ�
	dh = (float*)malloc(sizeof(float) * N*N); generate_d(dh, data);//d���ڴ�����������ڴ�ռ�
	objh = (float*)malloc(sizeof(float) * POP); initial_obj(objh);//obj���ڴ�����������ڴ�ռ�

	hipMalloc((void**)&indexd, sizeof(int) * SIZE);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&Id1, sizeof(float) * POP*P*POP);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&Id2, sizeof(float) * N);//I2���Դ�����������Դ�ռ�
	hipMalloc((void**)&sd1, sizeof(float) * N*POP);//I1���ڴ�����������Դ�ռ�
	hipMalloc((void**)&sd2, sizeof(float) * P*POP);//I2���Դ�����������Դ�ռ�
	hipMalloc((void**)&td, sizeof(float) * SIZE);//t���Դ�����������Դ�ռ�
	hipMalloc((void**)&dd, sizeof(float) * N*N);//d���Դ�����������Դ�ռ�
	hipMalloc((void**)&objd, sizeof(float) * POP);//obj���Դ�����������Դ�ռ�
	hipMalloc((void**)&initialxd, sizeof(float) * SIZE);
	hipMemcpy(initialxd, xh, sizeof(float) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�
	hipMemcpy(initialvd, initialvh, sizeof(float) * SIZE, hipMemcpyHostToDevice); //�ڴ��е�x���Ƶ��Դ�

	hipMemcpy(indexd, indexh, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(Id1, Ih1, sizeof(float) * POP*P*POP, hipMemcpyHostToDevice);
	hipMemcpy(Id2, Ih2, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(td, th, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dd, dh, sizeof(float) * N*N, hipMemcpyHostToDevice);
	hipMemcpy(objd, objh, sizeof(float) * POP, hipMemcpyHostToDevice);



	float *yh, *yd, *gbestd, *gbesth, *gbestxd, *gbestxh, *lbesth, *lbestxh, *lbestd, *lbestxd;
	yh = (float*)malloc(sizeof(float) *MAXITER);
	gbesth = (float*)malloc(sizeof(float));
	gbestxh = (float*)malloc(sizeof(float)*N*P);
	lbestxh = (float*)malloc(sizeof(float)*SIZE);
	lbesth = (float*)malloc(sizeof(float)*POP);
	hipMalloc((void**)&yd, sizeof(float) * MAXITER);
	hipMalloc((void**)&gbestd, sizeof(float));
	hipMalloc((void**)&gbestxd, sizeof(float)*N*P);
	hipMalloc((void**)&lbestd, sizeof(float)*POP);
	hipMalloc((void**)&lbestxd, sizeof(float)*SIZE);

	float *rdh1, *rdh2;
	float *rdd1, *rdd2;
	rdh1 = (float*)malloc(sizeof(float) *POP);
	rdh2 = (float*)malloc(sizeof(float) *POP);
	hipMalloc((void**)&rdd1, sizeof(float) * POP);
	hipMalloc((void**)&rdd2, sizeof(float) * POP);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	srand((unsigned int)time(NULL));
	hiprandState* devStates;

	hipMalloc(&devStates, POP * sizeof(hiprandState));

	srand(time(0));
	int gen;
	int is_stoph, *is_stopd, stop_number = STOPNUM;
	hipMalloc((void**)&is_stopd, sizeof(int));
	for (gen = 0; gen < 500; gen++)
	{



		generate_s1(handle, sd1, initialxd, Id1);
		generate_s2(handle, sd2, initialxd, Id2);
		generate_t(handle, td, dd, initialxd);
		//�������ӹ��࣬�����Ż���
		setup_kernel << <BLOCKS, THREAD_NUM >> > (devStates, rand());
		parallel_updated_GPU << <BLOCKS, THREAD_NUM >> > (objd, lbestd, lbestxd, initialxd, xd, dd, sd1, sd2, td, indexd, gen, devStates);//����ÿ��
		generate_global_best << <1, 1, 0 >> > (yd, gbestd, gbestxd, lbestd, lbestxd, gen);
		generate_rd1_rd2 << <RD_BLOCKS, RD_THREAD_NUM >> > (rdd1, rdd2, devStates);
		//hipMemcpy(yh, yd, sizeof(float)*MAXITER, hipMemcpyDeviceToHost);
		pso << <PSO_BLOCKS, PSO_THREAD_NUM >> > (xd, initialxd, initialvd, gbestxd, lbestxd, rdd1, rdd2, rand());//����ÿ��
		if (gen > stop_number)
		{
			stop << <1, 1, 0 >> > (yd, is_stopd, gen);
			hipMemcpy(&is_stoph, is_stopd, sizeof(int), hipMemcpyDeviceToHost);
			if (is_stoph == 1)
				break;
			else
				stop_number = stop_number + STOPNUM;
		}
	}

	hipMemcpy(gbestxh, gbestxd, sizeof(float) * N*P, hipMemcpyDeviceToHost);
	hipMemcpy(yh, yd, sizeof(float)*MAXITER, hipMemcpyDeviceToHost);





	//�ͷ��ڴ�
	free(indexh);
	free(xh); free(xth);
	free(sh1); free(sh2);
	free(Ih1); free(Ih2);
	free(th); free(dh);
	//�ͷ��Դ�
	hipFree(indexd);
	hipFree(xd); hipFree(xtd);
	hipFree(sd1); hipFree(sd2);
	hipFree(Id1); hipFree(Id2);
	hipFree(td); hipFree(dd);
	//�ͷž��
	hipblasDestroy(handle);

}